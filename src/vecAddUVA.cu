#define N 256

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vecAdd(float *a, float *b, float *c){
    c[threadIdx.x] = a[threadIdx.x]+b[threadIdx.x];
}

int main(void){

    // number of bytes to alloc for arrays    
    size_t numBytes = N*sizeof(float);

    // init host and device pointers
    float *ha, *hb, *hc;

    // alloc host memory/arrays (pinned, mapped)
    hipHostAlloc(&ha,numBytes,hipHostMallocMapped);
    hipHostAlloc(&hb,numBytes,hipHostMallocMapped);
    hipHostAlloc(&hc,numBytes,hipHostMallocMapped);

    // init host arrays
    for(int i=0; i<N; i++){ ha[i]=1.0; hb[i]=1.0; }

    // launch configuration
    dim3 gridSz(1,1,1), blockSz(N,1,1);

    // launch CUDA kernel
    vecAdd<<<gridSz,blockSz>>>(ha,hb,hc);

    // wait for kernel to finish
    hipDeviceSynchronize();

    // kernel result (no memcpy!)
    for (int i=1; i<N; i++){ printf("c[%d]: %f\n",i,hc[i]);}

    // free host memory
    hipHostFree(ha);  hipHostFree(hb);  hipHostFree(hc);
}
