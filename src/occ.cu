int arraySize = 1024*1024;
int blockSize, minGridSize, gridSize, maxActiveBlocks;

hipOccupancyMaxPotentialBlockSize( 
  &minGridSize, &blockSize, MyKernel, 0, 0);
 
gridSize = (arraySize + blockSize - 1) / blockSize; 

// MyKerel<<<gridSize,blockSize>>>(args);

hipOccupancyMaxActiveBlocksPerMultiprocessor( 
  &maxActiveBlocks, MyKernel, blockSize, 0);
  
int dev; hipDeviceProp_t p;
hipGetDevice(&dev); 
hipGetDeviceProperties(&p, dev);

// calculate theoretical occupancy
float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize);
