#define N 256

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vecAdd(float *a, float *b, float *c){
    c[threadIdx.x] = a[threadIdx.x]+b[threadIdx.x];
}

int main(void){

    // number of bytes to alloc for arrays    
    size_t numBytes = N*sizeof(float);

    // init host and device pointers
    float *ha, *hb, *hc, *da, *db, *dc;

    // alloc host memory/arrays (pinned, mapped)
    hipHostAlloc(&ha,numBytes,hipHostMallocMapped);
    hipHostAlloc(&hb,numBytes,hipHostMallocMapped);
    hipHostAlloc(&hc,numBytes,hipHostMallocMapped);

    // init host arrays
    for(int i=0; i<N; i++){ ha[i]=1.0; hb[i]=1.0; }

    // get device pointers
    hipHostGetDevicePointer(&da, ha,0);
    hipHostGetDevicePointer(&db, hb,0);
    hipHostGetDevicePointer(&dc, hc,0);

    // launch configuration
    dim3 gridSz(1,1,1), blockSz(N,1,1);

    // launch CUDA kernel
    vecAdd<<<gridSz,blockSz>>>(da,db,dc);

    // wait for kernel to finish
    hipDeviceSynchronize();

    // kernel result (no memcpy!)
    for (int i=1; i<N; i++){ printf("c[%d]: %f\n",i,hc[i]);}

    // free device pointers with cudaFree
    hipFree(da);  hipFree(db);  hipFree(dc);

    // free host pointers with cudaFreeHost
    hipHostFree(ha);  hipHostFree(hb);  hipHostFree(hc);
}
