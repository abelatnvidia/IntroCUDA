#include<stdio.h>
#include<hip/hip_runtime.h>

int main(void){    
    int deviceCount;  
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    hipGetDeviceProperties(&deviceProp,0);
    printf("There are %d gpu devices\n",deviceCount);
    printf("Device %s has %f GB of global memory\n",
        deviceProp.name,
        deviceProp.totalGlobalMem/pow(1024.0,3)
    );
}
