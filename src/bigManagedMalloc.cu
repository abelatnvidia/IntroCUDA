
#include <hip/hip_runtime.h>
#include<stdio.h>

int main(void){

  // init pointer to data
  char *data;
  
  // specify 32 GB of memory in bytes
  size_t numBytes = 1024*1024*1024;
  
  // Allocate 32 GB
  hipError_t err = hipMallocManaged(&data, numBytes/2);

  // blab about it
  printf("malloc status: %s\n",hipGetErrorString(err));   

  return 0;
}
