#include<stdio.h>
#include<hip/hip_runtime.h>

int main(void){
    int deviceCount; 
    char deviceName[256];
    hipDevice_t device;
    size_t szMem; int szProc;
    hipInit(0);
    hipGetDeviceCount(&deviceCount);
    hipDeviceGet(&device,0);
    hipDeviceGetName(deviceName,255,device);
    hipDeviceTotalMem(&szMem,device);
    hipDeviceGetAttribute(&szProc,hipDeviceAttributeMultiprocessorCount,device);
    printf("There are %d devices detected\n",deviceCount);
    printf("Device %s has %f GB of global memory\n",
        deviceName,szMem/pow(1024.0,3));
    printf("Device multiprocessor count: %d\n",szProc);
}
