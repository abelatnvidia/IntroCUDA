#define N 256

#include <hip/hip_runtime.h>
#include<stdlib.h>

__global__ void vecAdd(float *a, float *b, float *c){
    c[threadIdx.x] = a[threadIdx.x]+b[threadIdx.x];
}

int main(void){

    // number of bytes to alloc for arrays    
    size_t numBytes = N*sizeof(float);

    // init host and device pointers
    float *ha, *hb, *hc, *da, *db, *dc;

    // alloc host memory/arrays (pagable memory)
    ha = (float*)malloc(numBytes);
    hb = (float*)malloc(numBytes);
    hc = (float*)malloc(numBytes);

    // mem alloc arrays on the GPU device
    hipMalloc(&da,numBytes);
    hipMalloc(&db,numBytes);
    hipMalloc(&dc,numBytes);

    // copy host arrays to device
    hipMemcpy(da, ha, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(dc, hc, numBytes, hipMemcpyHostToDevice);

    // launch configuration
    dim3 gridSz (1,1,1), blockSz(N,1,1);

    // launch CUDA kernel
    vecAdd<<<gridSz,blockSz>>>(da,db,dc);

    // wait for kernel to finish
    hipDeviceSynchronize();

    // free host memory
    free(ha);  free(hb);  free(hc);

    // free device memory
    hipFree(da);  hipFree(db);  hipFree(dc);
}
