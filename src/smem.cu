
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void myKernel()
{ 
    __shared__ float sdata[1024*1024*1024]; 
    sdata[blockIdx.x] = blockIdx.x;
}
int main(void){ 
    myKernel<<<100,1>>>(); 
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    printf("status: %s\n",hipGetErrorString(err));
    return 0; 
}

